#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

const int BLOCK_NUM = 4;
const int THREAD_PER_BLOCK = 512;
const int THREAD_NUM = BLOCK_NUM * THREAD_PER_BLOCK;

//kerbel function of matrix multiplication
template<class T>
__global__ void gpuMatMultKernel(T *a, T *b, T *result, const int N, const int M, const int K) {
	//thread id
	int tid = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x
		+ blockIdx.x * blockDim.x + threadIdx.x;
	while (tid < N * K) {
		int row = tid / N;
		int col = tid % K;
		T temp = 0;
		for (int i = 0; i < M; ++i) {
			temp += a[row * M + i] * b[i * K + col];
		}
		result[tid] = temp;
		tid += THREAD_NUM;
	}
}

//show the information of all devices on this computer
void showDevice() {
	hipDeviceProp_t deviceProp;
	int deviceCount = 0;
	hipError_t hipError_t;
	hipError_t = hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("û�м�⵽�豸");
		return;
	}
	for (int i = 0; i < deviceCount; i++)
	{
		hipError_t = hipGetDeviceProperties(&deviceProp, i);

		printf("�豸 %d ����Ҫ����:\n", i);
		printf("�豸�Կ��ͺţ� %s\n", deviceProp.name);
		printf("�豸ȫ���ڴ�����(MB)�� %d\n", deviceProp.totalGlobalMem / 1024 / 1024);
		printf("�豸��һ���߳̿飨Block���п��õ�������ڴ�(KB)�� %d\n", deviceProp.sharedMemPerBlock / 1024);
		printf("�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� %d\n", deviceProp.regsPerBlock);
		printf("�豸��һ���߳̿飨Block���ɰ���������߳�������%d\n", deviceProp.maxThreadsPerBlock);
		printf("�豸�ļ��㹦�ܼ���Compute Capability���İ汾�ţ�%d.%d\n", deviceProp.major, deviceProp.minor);
		printf("�豸�϶ദ������������%d\n", deviceProp.multiProcessorCount);
	}
}

//Matrix Multiplication
//a[n][m] * b[m][k] = c[n][k]
extern void matrixMultOnGPU(int **a, int **b, int **c, int n, int m, int k) {
	//printf("starting CUDA matrix multiplication (int)\n");

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("Error: No device");
		return;
	}

	//parameters on device
	int *dev_a = NULL;
	int *dev_b = NULL;
	int *dev_c = NULL;

	//multi-device is not supported
	hipSetDevice(0);

	hipError_t error;

	//allocate memory for calculations on device
	hipMalloc((void**)&dev_a, sizeof(int) * n * m);
	hipMalloc((void**)&dev_b, sizeof(int) * m * k);
	hipMalloc((void**)&dev_c, sizeof(int) * n * k);
	error = hipGetLastError();
	printf("CUDA malloc: %s\n", hipGetErrorString(error));

	//copy memory to device
	hipMemcpy(dev_a, a[0], sizeof(int) * n * m, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b[0], sizeof(int) * m * k, hipMemcpyHostToDevice);
	error = hipGetLastError();
	printf("CUDA host to device: %s\n", hipGetErrorString(error));

	time_t t = clock();
	//kernel function <<<block_number, thread_per_block>>>
	gpuMatMultKernel<<<BLOCK_NUM, THREAD_PER_BLOCK>>>(dev_a, dev_b, dev_c, n, m, k);
	hipDeviceSynchronize(); //wait for multiplication over
	printf("%dms\n", clock() - t);
	error = hipGetLastError();
	printf("CUDA multi: %s\n", hipGetErrorString(error));

	//get memory from device
	hipMemcpy(&c[0][0], dev_c, n * k * sizeof(int), hipMemcpyDeviceToHost);
	error = hipGetLastError();
	printf("CUDA device to host: %s\n", hipGetErrorString(error));

	//free
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	error = hipGetLastError();
	printf("CUDA mem free: %s\n", hipGetErrorString(error));

	//printf("CUDA matrix multiplication over\n");
}

//float version of matrix multiplication
extern void matrixMultOnGPU(float **a, float **b, float **c, int n, int m, int k) {
	//printf("starting CUDA matrix multiplication (float)\n");

	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("Error: No device");
		return;
	}

	//parameters on device
	float *dev_a = NULL;
	float *dev_b = NULL;
	float *dev_c = NULL;

	//multi-device is not supported
	hipSetDevice(0);

	hipError_t error;

	//allocate memory for calculations on device
	hipMalloc((void**)&dev_a, sizeof(float) * n * m);
	hipMalloc((void**)&dev_b, sizeof(float) * m * k);
	hipMalloc((void**)&dev_c, sizeof(float) * n * k);
	error = hipGetLastError();
	printf("CUDA malloc: %s\n", hipGetErrorString(error));

	//copy memory to device
	hipMemcpy(dev_a, a[0], sizeof(float) * n * m, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b[0], sizeof(float) * m * k, hipMemcpyHostToDevice);
	error = hipGetLastError();
	printf("CUDA host to device: %s\n", hipGetErrorString(error));

	//kernel function <<<block_number, thread_per_block>>>
	time_t t = clock();
	gpuMatMultKernel << <BLOCK_NUM, THREAD_PER_BLOCK >> > (dev_a, dev_b, dev_c, n, m, k);
	hipDeviceSynchronize(); //wait for multiplication over
	printf("%dms\n", clock() - t);
	error = hipGetLastError();
	printf("CUDA multi: %s\n", hipGetErrorString(error));

	//get memory from device
	hipMemcpy(c[0], dev_c, n * k * sizeof(float), hipMemcpyDeviceToHost);
	error = hipGetLastError();
	printf("CUDA device to host: %s\n", hipGetErrorString(error));

	//free
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	error = hipGetLastError();
	printf("CUDA mem free: %s\n", hipGetErrorString(error));

	//printf("CUDA matrix multiplication over\n");
}
